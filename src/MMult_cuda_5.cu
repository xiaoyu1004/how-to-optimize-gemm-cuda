#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>

// CUDA runtime
#include "helper.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

// MY_MMult = [
// 1024 6467.51 7.247925e-05
// 2048 6693.74 1.525879e-04
// 3072 7096.70 2.288818e-04
// 4096 6677.67 4.425049e-04
// ];
/**
 * 和 version4 的区别：
 * 1. 修改了分块尺寸
 * 2. 每个 block 有 8x8 个线程，每个线程计算 4x4 个结果
 */
template <int BLOCK, int STRIDE>
__global__ void sgemm(int m, int n, int k, float *a, int lda, float *b, int ldb,
                      float *c, int ldc) {
  // blockIdx control subpanel matrix
  constexpr int STEP = BLOCK * STRIDE;
  const int tx = threadIdx.x * STRIDE;
  const int ty = threadIdx.y * STRIDE;
  const int bx = blockIdx.x * STEP;
  const int by = blockIdx.y * STEP;

  float *begin_a = a + by * k;
  float *begin_b = b + bx;
  float *end_a = begin_a + k;

  float sum[STRIDE][STRIDE] = {0.f};
  for (float *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
       a_ptr += STEP, b_ptr += STEP * n) {
    __shared__ __align__(16 * 1024) float ashare[STEP][STEP];
    __shared__ __align__(16 * 1024) float bshare[STEP][STEP];

    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        ashare[ty + i][tx + j] = a_ptr[(ty + i) * k + tx + j];
        bshare[ty + i][tx + j] = b_ptr[(ty + i) * n + tx + j];
      }
    }
    __syncthreads();

    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        for (int kk = 0; kk < STEP; ++kk) {
          sum[i][j] += ashare[ty + i][kk] * bshare[kk][tx + j];
        }
      }
    }

    __syncthreads();
  }

#pragma unroll
  for (int i = 0; i < STRIDE; ++i) {
    for (int j = 0; j < STRIDE; ++j) {
      c[(by + ty + i) * n + bx + tx + j] = sum[i][j];
    }
  }
}

void MY_MMult(hipblasHandle_t handle, int m, int n, int k, float *d_A, int lda,
              float *d_B, int ldb, float *d_C, int ldc) {

  constexpr int BLOCK = 8;
  constexpr int STRIDE = 4; // every thread calc STRIDExSTRIDE result
  dim3 block(BLOCK, BLOCK);
  dim3 grid((m + BLOCK - 1) / BLOCK / STRIDE, (n + BLOCK - 1) / BLOCK / STRIDE);

  sgemm<BLOCK, STRIDE><<<grid, block>>>(m, n, k, d_A, lda, d_B, ldb, d_C, ldc);
}
