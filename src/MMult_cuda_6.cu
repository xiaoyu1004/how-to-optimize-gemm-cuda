#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>

// CUDA runtime
#include "helper.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

// a = mxk, b = kxn
template <int BLOCK, int STRIDE>
__global__ void sgemm(int m, int n, int k, float *a, int lda, float *b, int ldb,
                      float *c, int ldc) {
  // blockIdx control subpanel matrix
  constexpr int STEP = BLOCK * STRIDE;
  const int tx = threadIdx.x * STRIDE;
  const int ty = threadIdx.y * STRIDE;
  const int bx = blockIdx.x * STEP;
  const int by = blockIdx.y * STEP;

  float *begin_a = a + by * k;
  float *begin_b = b + bx;
  float *end_a = begin_a + k;

  float sum[STRIDE][STRIDE] = {0.f};

  __shared__ float ashare[STEP][2 * STEP];
  __shared__ float bshare[2 * STEP][STEP];
  // bigger split
  for (float *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
       a_ptr += 2 * STEP, b_ptr += 2 * STEP * n) {

    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        ashare[ty + i][tx + j] = a_ptr[(ty + i) * k + tx + j];
        ashare[ty + i][tx + j + STEP] = a_ptr[(ty + i) * k + tx + j + STEP];

        bshare[ty + i][tx + j] = b_ptr[(ty + i) * n + tx + j];
        bshare[ty + i + STEP][tx + j] = b_ptr[(ty + i + STEP) * n + tx + j];
      }
    }
    __syncthreads();

    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        for (int kk = 0; kk < 2 * STEP; ++kk) {
          sum[i][j] += ashare[ty + i][kk] * bshare[kk][tx + j];
        }
      }
    }

    __syncthreads();
  }

#pragma unroll
  for (int i = 0; i < STRIDE; ++i) {
    for (int j = 0; j < STRIDE; ++j) {
      c[(by + ty + i) * n + bx + tx + j] = sum[i][j];
    }
  }
}

void MY_MMult(hipblasHandle_t handle, int m, int n, int k, float *d_A, int lda,
              float *d_B, int ldb, float *d_C, int ldc) {

  constexpr int BLOCK = 16;
  constexpr int STRIDE = 2; // every thread calc STRIDExSTRIDE result
  dim3 block(BLOCK, BLOCK);
  dim3 grid((m + BLOCK - 1) / BLOCK / STRIDE, (n + BLOCK - 1) / BLOCK / STRIDE);

  sgemm<BLOCK, STRIDE><<<grid, block>>>(m, n, k, d_A, lda, d_B, ldb, d_C, ldc);
}
